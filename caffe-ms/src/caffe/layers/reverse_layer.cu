#include "hip/hip_runtime.h"
#include "caffe/common_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void reverse_gpu(const int nthreads, const Dtype* from_data, Dtype* to_data, 
	const int* counts, const int axis_count, const int axis) {
  CUDA_KERNEL_LOOP(index, nthreads) {
  	int ind=(index/counts[axis])%axis_count;
  	int to_index=counts[axis]*(axis_count-2*ind-1)+index;
  	*(to_data+to_index)=*(from_data+index);
  }
}

template <typename Dtype>
void ReverseLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom, 
		const vector<Blob<Dtype>*>& top) {
	const int nthreads=bottom[0]->count();
	reverse_gpu<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
        nthreads, bottom[0]->gpu_data(), top[0]->mutable_gpu_data(), 
        bottom_counts_.gpu_data(), bottom[0]->shape(axis_), axis_);
}

template <typename Dtype>
void ReverseLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    	const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
	if (!propagate_down[0]) {
		return;
	}
	const int nthreads=bottom[0]->count();
	reverse_gpu<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
        nthreads, top[0]->gpu_diff(), bottom[0]->mutable_gpu_diff(), 
        bottom_counts_.gpu_data(), bottom[0]->shape(axis_), axis_);
}

INSTANTIATE_LAYER_GPU_FUNCS(ReverseLayer);

}  // namespace caffe
